#include "hip/hip_runtime.h"
#include <stdio.h>
#include <windows.h>
#include <time.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <iostream>

#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <hip/device_functions.h>
#include <sm_20_atomic_functions.h>

using namespace cv;

#define SPLIT_SIZE_X 32
#define SPLIT_SIZE_Y 24
#define SPLIT_SIZE_X2 32
#define SPLIT_SIZE_Y2 24
#define BLOCK_SIZE_X 36
#define BLOCK_SIZE_Y 28

#define IS_NOT_EDGE(a) (a < min_val)
#define IS_STRONG_EDGE(a) (a >= max_val)
#define IS_WEAK_EDGE(a)   (a >= min_val && a < max_val)

/*canny using cuda*/
void CUDA_Canny();
__global__ void CUDA_GaussianAndSobel(unsigned char* img, int width, int height, unsigned char* output_sobel, short* output_gradient);
__device__ void CUDA_Gaussian(unsigned char* img, int width, int height, int idx, unsigned char* output);
__device__ void CUDA_Sobel(unsigned char* img, int width, int height, int idx, unsigned char* output_sobel, short* gradient);
__global__ void CUDA_NonMaxSuppress(unsigned char* sobel, int width, int height, short* gradient, unsigned char* output);
__global__ void CUDA_DoubleThreshold(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* canny);
__device__ unsigned char CUDA_GetPixelVal(unsigned char* img, int width, int height, int i, int j);
__device__ short GetGradientDirection(int sobel_x, int sobel_y);
__device__ void CheckAndConvert(unsigned char* img, int width, int height, int i, int j, int min_val, int max_val, 
	                            unsigned short* weak_stack, unsigned short* stack_index, unsigned char* canny);
__global__ void CUDA_DoubleThreshold2(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* canny);


void DisplayGradient(short* gradient, int width, int height);
unsigned char GetPixelVal(unsigned char* img, int width, int height, int i, int j);
void NonMaxSuppress(unsigned char* sobel, int width, int height, short* gradient, unsigned char* output);
void DoubleThreshold(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* output);
void IsWeakEdge(unsigned char* sobel, int width, int height, int min_val, int max_val, int i, int j, unsigned short* stack, unsigned short* top, unsigned char* output);

__device__ void CUDA_SubDoubleThreshold(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* output);
__device__ void CUDA_IsWeakEdge(unsigned char* sobel, int width, int height, int min_val, int max_val, int i, int j, unsigned short* stack, unsigned short* top, unsigned char* output);


int main(void)
{
	printf("CANNY_CUDA\n");
	CUDA_Canny();
	//system("pause");
	return 0;
}

void CUDA_Canny()
{
	int width = 640;
	int height = 480;
	dim3 block_size_extended(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 block_size_normal(SPLIT_SIZE_X, SPLIT_SIZE_Y);
	dim3 block_size_normal2(SPLIT_SIZE_X2, SPLIT_SIZE_Y2);
	dim3 grid_size(width / SPLIT_SIZE_X, height / SPLIT_SIZE_Y);
	dim3 grid_size2(width / SPLIT_SIZE_X2, height / SPLIT_SIZE_Y2);
	Mat img_src, img_sobel, img_gradient, img_canny;

	VideoCapture camera(1);

	/*cpu memory*/
	unsigned char* cpu_img = new unsigned char[width * height];
	unsigned char* cpu_img2 = new unsigned char[width * height];
	unsigned char* cpu_img3 = new unsigned char[width * height];
	short* cpu_gradient = new short[width * height];

	/*gpu memory*/
	unsigned char* gpu_img;
	hipMalloc(&gpu_img, width * height * sizeof(unsigned char));
	unsigned char* gpu_sobel;
	hipMalloc(&gpu_sobel, width * height * sizeof(unsigned char));
	short* gpu_gradient;
	hipMalloc(&gpu_gradient, width * height * sizeof(short));
	unsigned char* gpu_canny;
	hipMalloc(&gpu_canny, width * height * sizeof(unsigned char));

	StopWatchInterface * timer_cublas;

	while (1)
	{
		camera >> img_src;
		//img_src = imread("F:/img_src/lena.jpg");
		resize(img_src, img_src, Size(width, height), 0, 0);
		cvtColor(img_src, img_src, CV_BGR2GRAY);
		//imshow("img_src", img_src);

		sdkCreateTimer(&timer_cublas);
		sdkStartTimer(&timer_cublas);

		/*1.copy to gpu memory*/
		hipMemcpy(gpu_img, img_src.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

		/*2.gauss filter*/
		CUDA_GaussianAndSobel << <grid_size, block_size_extended >> > (gpu_img, width, height, gpu_sobel, gpu_gradient);
		hipDeviceSynchronize();

		/*3.none max suppress*/
		CUDA_NonMaxSuppress << <grid_size, block_size_normal >> > (gpu_sobel, width, height, gpu_gradient, gpu_sobel);

		/*4.double threshold*/
		//CUDA_DoubleThreshold << <grid_size2, block_size_normal2 >> > (gpu_sobel, width, height, 40, 90, gpu_canny);
		CUDA_DoubleThreshold2 << <grid_size2, dim3(1,1) >> > (gpu_sobel, width, height, 40, 90, gpu_canny);

		/*copy to cpu memory*/
		hipMemcpy(cpu_img, gpu_canny, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipMemcpy(cpu_img2, gpu_sobel, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

		//img_canny = Mat(Size(width, height), CV_8UC1, cpu_img);
		//resize(img_canny, img_canny, Size(640, 480), 0, 0);
		//imshow("img_canny", img_canny);

		//DoubleThreshold(cpu_img2, width, height, 50, 90, cpu_img3);

		hipDeviceSynchronize();
		sdkStopTimer(&timer_cublas);
		printf("\ngpu_done %.3f(ms)\n\n", sdkGetTimerValue(&timer_cublas));

		img_sobel = Mat(Size(width, height), CV_8UC1, cpu_img);
		resize(img_sobel, img_sobel, Size(640, 480), 0, 0);
		imshow("img_sobel", img_sobel);

		if ('q' == waitKey(1))
		{
			destroyAllWindows();
			free(cpu_img);
			cpu_img = NULL;
			free(cpu_img2);
			cpu_img2 = NULL;
			free(cpu_img3);
			cpu_img3 = NULL;
			free(cpu_gradient);
			cpu_gradient = NULL;
			hipFree(gpu_img);
			hipFree(gpu_sobel);
			hipFree(gpu_gradient);
			hipFree(gpu_canny);

			break;
		}
		
	}
}

__global__ void CUDA_GaussianAndSobel(unsigned char* img, int width, int height, unsigned char* output_sobel, short* output_gradient)
{
	__shared__ unsigned char cache[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];
	__shared__ unsigned char gauss[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];
	__shared__ unsigned char sobel[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];
	short gradient = 0;

	/*alloct img to cache*/
	int raw_index = SPLIT_SIZE_X * SPLIT_SIZE_Y * blockIdx.y * gridDim.x + blockIdx.x * SPLIT_SIZE_X + SPLIT_SIZE_X * gridDim.x * threadIdx.y + threadIdx.x;
	int pixel_val = CUDA_GetPixelVal(img, width, height, raw_index / width - 2, raw_index % width - 2);
	int cache_index = blockDim.x * threadIdx.y + threadIdx.x;
	cache[cache_index] = pixel_val;
	__syncthreads();

	/*gauss filter*/
	CUDA_Gaussian(cache, blockDim.x, blockDim.y, cache_index, gauss);
	__syncthreads();

	/*sobel filter*/
	CUDA_Sobel(gauss, blockDim.x, blockDim.y, cache_index, sobel, &gradient);

	/*cute edge*/
	if (threadIdx.y <= 1 || threadIdx.y >= blockDim.y - 2 || 
		threadIdx.x <= 1 || threadIdx.x >= blockDim.x - 2)
		return;
	int new_id = blockIdx.y * SPLIT_SIZE_X * SPLIT_SIZE_Y * gridDim.x + (threadIdx.y - 2) * SPLIT_SIZE_X * gridDim.x + blockIdx.x * SPLIT_SIZE_X + (threadIdx.x - 2);

	/*store result*/
	output_gradient[new_id] = gradient;
	output_sobel[new_id] = sobel[cache_index];
}

__device__ void CUDA_Gaussian(unsigned char* img, int width, int height, int idx, unsigned char* output)
{
	int new_pixel_value = 0;
	new_pixel_value = CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x - 1) * 0.07511 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x    ) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x + 1) * 0.07511 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y    , threadIdx.x - 1) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y    , threadIdx.x    ) * 0.20418 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y    , threadIdx.x + 1) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x - 1) * 0.07511 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x    ) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x + 1) * 0.07511;
	output[idx] = new_pixel_value;
}

__device__ void CUDA_Sobel(unsigned char* img, int width, int height, int idx, unsigned char* output_sobel, short* gradient)
{
	int sobel_x = 0;
	int sobel_y = 0;
	int sobel = 0;
	sobel_x = CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x - 1) * (1) +
			  CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x    ) * (2) +
			  CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x + 1) * (1) +
			  CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x - 1) * (-1) +
			  CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x    ) * (-2) +
			  CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x + 1) * (-1);
	sobel_y = CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x - 1) * (-1) +
		      CUDA_GetPixelVal(img, width, height, threadIdx.y - 1, threadIdx.x + 1) * (1) +
		      CUDA_GetPixelVal(img, width, height, threadIdx.y    , threadIdx.x - 1) * (-2) +
		      CUDA_GetPixelVal(img, width, height, threadIdx.y    , threadIdx.x + 1) * (2) +
		      CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x - 1) * (-1) +
		      CUDA_GetPixelVal(img, width, height, threadIdx.y + 1, threadIdx.x + 1) * (1);
	sobel = sqrtf((float)(sobel_x * sobel_x + sobel_y * sobel_y));
	sobel = sobel > 255 ? 255 : sobel;

	output_sobel[idx] = sobel;
	*gradient = GetGradientDirection(sobel_x, sobel_y);
}

__global__ void CUDA_NonMaxSuppress(unsigned char* sobel, int width, int height, short* gradient, unsigned char* output)
{
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (id >= width * height)
		return;
	int i = id / width;
	int j = id % width;
	float weight = 0;
	int g0, g1, g2, g3;
	int temp_gradient = gradient[id] < 0 ? gradient[id] + 180 : gradient[id];
	if (temp_gradient >= 0 && temp_gradient < 45)
	{
		weight = temp_gradient / 45.0;
		g0 = CUDA_GetPixelVal(sobel, width, height, i    , j + 1);
		g1 = CUDA_GetPixelVal(sobel, width, height, i - 1, j + 1);
		g2 = CUDA_GetPixelVal(sobel, width, height, i    , j - 1);
		g3 = CUDA_GetPixelVal(sobel, width, height, i + 1, j - 1);
	}
	else if (temp_gradient >= 45 && temp_gradient < 90)
	{
		weight = (90 - temp_gradient) / 45.0;
		g0 = CUDA_GetPixelVal(sobel, width, height, i - 1, j    );
		g1 = CUDA_GetPixelVal(sobel, width, height, i - 1, j + 1);
		g2 = CUDA_GetPixelVal(sobel, width, height, i + 1, j    );
		g3 = CUDA_GetPixelVal(sobel, width, height, i + 1, j - 1);
	}
	else if (temp_gradient >= 90 && temp_gradient < 135)
	{
		weight = (temp_gradient - 90) / 45.0;
		g0 = CUDA_GetPixelVal(sobel, width, height, i - 1, j    );
		g1 = CUDA_GetPixelVal(sobel, width, height, i - 1, j - 1);
		g2 = CUDA_GetPixelVal(sobel, width, height, i + 1, j    );
		g3 = CUDA_GetPixelVal(sobel, width, height, i + 1, j + 1);
	}
	else if (temp_gradient >= 135 && temp_gradient <= 180)
	{
		weight = (180 - temp_gradient) / 45.0;
		g0 = CUDA_GetPixelVal(sobel, width, height, i    , j - 1);
		g1 = CUDA_GetPixelVal(sobel, width, height, i - 1, j - 1);
		g2 = CUDA_GetPixelVal(sobel, width, height, i    , j + 1);
		g3 = CUDA_GetPixelVal(sobel, width, height, i + 1, j + 1);
	}
	int dot1 = g0 * (1 - weight) + g1 * weight;
	int dot2 = g2 * (1 - weight) + g3 * weight;
	if (sobel[id] >= dot1 && sobel[id] >= dot2)
		output[id] = sobel[id];
	else
		output[id] = 0;
}

__global__ void CUDA_DoubleThreshold(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* canny)
{
	__shared__ unsigned char cache[SPLIT_SIZE_X2 * SPLIT_SIZE_Y2];
	__shared__ unsigned char cache2[SPLIT_SIZE_X2 * SPLIT_SIZE_Y2];
	int raw_index = blockDim.x * blockDim.y * blockIdx.y * gridDim.x + blockIdx.x * blockDim.x + blockDim.x * gridDim.x * threadIdx.y + threadIdx.x;
	int pixel_val = CUDA_GetPixelVal(sobel, width, height, raw_index / width, raw_index % width);
	int cache_index = blockDim.x * threadIdx.y + threadIdx.x;
	cache[cache_index] = pixel_val;
	__syncthreads();
	
	if (IS_STRONG_EDGE(cache[cache_index]))
	{
		unsigned short weak_stack[SPLIT_SIZE_X2 * SPLIT_SIZE_Y2];
		unsigned short stack_cnt = 0;
		CheckAndConvert(cache, blockDim.x, blockDim.y, threadIdx.y, threadIdx.x, min_val, max_val, weak_stack, &stack_cnt, cache2);
		unsigned short center_index = 0;
		while(stack_cnt > 0)
		{
			center_index = weak_stack[stack_cnt - 1];
			stack_cnt--;
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x - 1, center_index % blockDim.x - 1, min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x - 1, center_index % blockDim.x    , min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x - 1, center_index % blockDim.x + 1, min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x    , center_index % blockDim.x - 1, min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x    , center_index % blockDim.x + 1, min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x + 1, center_index % blockDim.x - 1, min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x + 1, center_index % blockDim.x    , min_val, max_val, weak_stack, &stack_cnt, cache2);
			CheckAndConvert(cache, blockDim.x, blockDim.y, center_index / blockDim.x + 1, center_index % blockDim.x + 1, min_val, max_val, weak_stack, &stack_cnt, cache2);
			__syncthreads();
		}
	}
	else if (IS_NOT_EDGE(cache[cache_index]))
	{
		cache2[cache_index] = 0;
	}

	int new_id = blockIdx.y * blockDim.x * blockDim.y * gridDim.x + threadIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	canny[new_id] = cache2[cache_index];
	__syncthreads();
}

__device__ void CheckAndConvert(unsigned char* img, int width, int height, int i, int j, int min_val, int max_val, 
	                            unsigned short* weak_stack, unsigned short* stack_index, unsigned char* canny)
{
	int temp_index = width * i + j;
	if (IS_WEAK_EDGE(CUDA_GetPixelVal(img, width, height, i, j)) || 
		IS_STRONG_EDGE(CUDA_GetPixelVal(img, width, height, i, j)))
	{
		weak_stack[*stack_index] = temp_index;
		(*stack_index)++;
		canny[temp_index] = 255;
	}
	else
	{
		canny[temp_index] = 0;
	}
}

__global__ void CUDA_DoubleThreshold2(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* canny)
{
	__shared__ unsigned char cache[SPLIT_SIZE_X2 * SPLIT_SIZE_Y2];
	__shared__ unsigned char output[SPLIT_SIZE_X2 * SPLIT_SIZE_Y2];
	int raw_index = SPLIT_SIZE_X2 * SPLIT_SIZE_Y2 * blockIdx.y * gridDim.x + blockIdx.x * SPLIT_SIZE_X2 + SPLIT_SIZE_X2 * gridDim.x * threadIdx.y + threadIdx.x;
	for (int i = 0; i < SPLIT_SIZE_Y2; i++)
	{
		for (int j = 0; j < SPLIT_SIZE_X2; j++)
		{
			cache[i * SPLIT_SIZE_X2 + j] = CUDA_GetPixelVal(sobel, width, height, raw_index / width + i, raw_index % width + j);
		}
	}

	CUDA_SubDoubleThreshold(cache, SPLIT_SIZE_X2, SPLIT_SIZE_Y2, min_val, max_val, output);

	for (int i = 0; i < SPLIT_SIZE_Y2; i++)
	{
		for (int j = 0; j < SPLIT_SIZE_X2; j++)
		{
			int new_id = blockIdx.y * SPLIT_SIZE_X2 * SPLIT_SIZE_Y2 * gridDim.x + i * SPLIT_SIZE_X2 * gridDim.x + blockIdx.x * SPLIT_SIZE_X2 + j;
			canny[new_id] = output[i * SPLIT_SIZE_X2 + j];
		}
	}
}

__device__ unsigned char CUDA_GetPixelVal(unsigned char* img, int width, int height, int i, int j)
{
	if (i >= height || i < 0)
		return 0;
	else if (j >= width || j < 0)
		return 0;
	return *(img + i * width + j);
}

__device__ short GetGradientDirection(int sobel_x, int sobel_y)
{
	short gradient = (atan2f(sobel_x, sobel_y) / 3.1415926 * 180.0);
	//gradient = gradient < 0 ? gradient + 180 : gradient;
	return gradient;
}

void DisplayGradient(short* gradient, int width, int height)
{
	Mat img = Mat::zeros(Size(width, height), CV_8UC3);
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			if (abs(*(gradient + i * width + j)) >= 0 && abs(*(gradient + i * width + j)) < 45)
			{
				img.at<Vec3b>(i, j) = Vec3b(255, 0, 0);
			}
			else if (abs(*(gradient + i * width + j)) >= 45 && abs(*(gradient + i * width + j)) < 90)
			{
				img.at<Vec3b>(i, j) = Vec3b(0, 255, 0);
			}
			else if (abs(*(gradient + i * width + j)) >= 90 && abs(*(gradient + i * width + j)) < 135)
			{
				img.at<Vec3b>(i, j) = Vec3b(0, 0, 255);
			}
			else if (abs(*(gradient + i * width + j)) >= 135 && abs(*(gradient + i * width + j)) <= 180)
			{
				img.at<Vec3b>(i, j) = Vec3b(128, 128, 128);
			}
		}
	}
	imshow("gradient", img);
}

unsigned char GetPixelVal(unsigned char* img, int width, int height, int i, int j)
{
	if (i >= height || i < 0)
		return 0;
	else if (j >= width || j < 0)
		return 0;
	return *(img + i * width + j);
}

void NonMaxSuppress(unsigned char* sobel, int width, int height, short* gradient, unsigned char* output)
{
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			int id = i * width + j;
			float weight = 0;
			int g0, g1, g2, g3;
			int temp_gradient = gradient[id] < 0 ? gradient[id] + 180 : gradient[id];
			if (temp_gradient >= 0 && temp_gradient < 45)
			{
				weight = temp_gradient / 45.0;
				g0 = GetPixelVal(sobel, width, height, i, j + 1);
				g1 = GetPixelVal(sobel, width, height, i - 1, j + 1);
				g2 = GetPixelVal(sobel, width, height, i, j - 1);
				g3 = GetPixelVal(sobel, width, height, i + 1, j - 1);
			}
			else if (temp_gradient >= 45 && temp_gradient < 90)
			{
				weight = (90 - temp_gradient) / 45.0;
				g0 = GetPixelVal(sobel, width, height, i - 1, j);
				g1 = GetPixelVal(sobel, width, height, i - 1, j + 1);
				g2 = GetPixelVal(sobel, width, height, i + 1, j);
				g3 = GetPixelVal(sobel, width, height, i + 1, j - 1);
			}
			else if (temp_gradient >= 90 && temp_gradient < 135)
			{
				weight = (temp_gradient - 90) / 45.0;
				g0 = GetPixelVal(sobel, width, height, i - 1, j);
				g1 = GetPixelVal(sobel, width, height, i - 1, j - 1);
				g2 = GetPixelVal(sobel, width, height, i + 1, j);
				g3 = GetPixelVal(sobel, width, height, i + 1, j + 1);
			}
			else if (temp_gradient >= 135 && temp_gradient <= 180)
			{
				weight = (180 - temp_gradient) / 45.0;
				g0 = GetPixelVal(sobel, width, height, i, j - 1);
				g1 = GetPixelVal(sobel, width, height, i - 1, j - 1);
				g2 = GetPixelVal(sobel, width, height, i, j + 1);
				g3 = GetPixelVal(sobel, width, height, i + 1, j + 1);
			}
			int dot1 = g0 * (1 - weight) + g1 * weight;
			int dot2 = g2 * (1 - weight) + g3 * weight;
			if (sobel[id] > dot1 && sobel[id] > dot2)
				output[id] = sobel[id];
			else
				output[id] = 0;
		}
	}
}

void DoubleThreshold(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* output)
{
	unsigned short* weak_stack = new unsigned short[width * height];
	unsigned short stack_top = 0;
	unsigned short center_index = 0;

	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			if (IS_STRONG_EDGE(GetPixelVal(sobel, width, height, i, j)))
			{
				stack_top = 0;
				IsWeakEdge(sobel, width, height, min_val, max_val, i, j, weak_stack, &stack_top, output);
				while (stack_top > 0)
				{
					center_index = weak_stack[stack_top - 1];
					stack_top--;
					IsWeakEdge(sobel, width, height, min_val, max_val, i - 1, j - 1, weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i - 1, j    , weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i - 1, j + 1, weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i    , j - 1, weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i    , j + 1, weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i + 1, j - 1, weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i + 1, j    , weak_stack, &stack_top, output);
					IsWeakEdge(sobel, width, height, min_val, max_val, i + 1, j + 1, weak_stack, &stack_top, output);
				}
			}
			else if (IS_NOT_EDGE(GetPixelVal(sobel, width, height, i, j)))
			{
				output[i * width + j] = 0;
			}
		}
	}

	delete[] weak_stack;
	weak_stack = nullptr;
}

void IsWeakEdge(unsigned char* sobel, int width, int height, int min_val, int max_val, int i, int j, unsigned short* stack, unsigned short* top, unsigned char* output)
{
	if (IS_WEAK_EDGE(GetPixelVal(sobel, width, height, i, j)) ||
		IS_STRONG_EDGE(GetPixelVal(sobel, width, height, i, j)))
	{
		output[i * width + j] = 255;
		stack[*top] = i * width + j;
		*top++;
	}
	else
	{
		output[i * width + j] = 0;
	}
}

__device__ void CUDA_SubDoubleThreshold(unsigned char* sobel, int width, int height, int min_val, int max_val, unsigned char* output)
{
	unsigned short* weak_stack = new unsigned short[width * height];
	unsigned short stack_top = 0;
	unsigned short center_index = 0;

	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			if (IS_STRONG_EDGE(CUDA_GetPixelVal(sobel, width, height, i, j)))
			{
				stack_top = 0;
				CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i, j, weak_stack, &stack_top, output);
				while (stack_top > 0)
				{
					center_index = weak_stack[stack_top - 1];
					stack_top--;
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i - 1, j - 1, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i - 1, j, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i - 1, j + 1, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i, j - 1, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i, j + 1, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i + 1, j - 1, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i + 1, j, weak_stack, &stack_top, output);
					CUDA_IsWeakEdge(sobel, width, height, min_val, max_val, i + 1, j + 1, weak_stack, &stack_top, output);
					__syncthreads();
				}
			}
			else if (IS_NOT_EDGE(CUDA_GetPixelVal(sobel, width, height, i, j)))
			{
				output[i * width + j] = 0;
			}
		}
	}

	delete[] weak_stack;
	weak_stack = nullptr;
}

__device__ void CUDA_IsWeakEdge(unsigned char* sobel, int width, int height, int min_val, int max_val, int i, int j, unsigned short* stack, unsigned short* top, unsigned char* output)
{
	if (IS_WEAK_EDGE(CUDA_GetPixelVal(sobel, width, height, i, j)) ||
		IS_STRONG_EDGE(CUDA_GetPixelVal(sobel, width, height, i, j)))
	{
		output[i * width + j] = 255;
		stack[*top] = i * width + j;
		*top++;
	}
	else
	{
		output[i * width + j] = 0;
	}
}
