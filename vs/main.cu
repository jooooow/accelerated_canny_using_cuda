#include "hip/hip_runtime.h"
#include <stdio.h>
#include <windows.h>
#include <time.h>
#include <opencv2/opencv.hpp>
#include <math.h>

#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <hip/device_functions.h>

using namespace cv;

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 24

/*test1*/
void Test1();
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
		C[i] = A[i] * B[i];
}
void Addvec(float* A, float* B, float* C, int N)
{
	for (int i = 0; i < N; i++)
		C[i] = A[i] * B[i];
}

/*opencvtest*/
void OpencvTest();
void ConvCPU(unsigned char* img, float* kernel, int img_width, int img_height, int kernel_size, unsigned char* output);
__global__ void ConvGPU(unsigned char* img, float* kernel, int img_width, int img_height, int kernel_size, unsigned char* output);
__global__ void ReverseGPU(unsigned char* img, int img_width, int img_height);

/*cuda array sum*/
void GpuArraySum();
__global__ void ArraySum(int* arr, int* res, int size);



/*canny using cuda*/
void CUDA_Canny();
__global__ void CUDA_GaussianAndSobel(unsigned char* img, int width, int height, unsigned char* output);
__global__ void CUDA_GaussianAndSobel2(unsigned char* img, int width, int height, unsigned char* output);
__device__ void CUDA_Gaussian(unsigned char* img, int width, int height, int idx, unsigned char* output);
__device__ void CUDA_Sobel(unsigned char* img, int width, int height, int idx, unsigned char* output);
__device__ unsigned char CUDA_GetPixelVal(unsigned char* img, int width, int height, int i, int j);

__global__ void CUDA_AssignTest(int* arr);


int main(void)
{
	//Test1();
	//OpencvTest();
	//GpuArraySum();
	CUDA_Canny();

	//system("pause");
	return 0;
}


void CUDA_Canny()
{
	printf("***CUDA_CANNY***\n");
	
	int width = 640;
	int height = 480;
	dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 grid_size(width / BLOCK_SIZE_X, height / BLOCK_SIZE_Y);
	Mat img_src, img_sobel, img_canny;

	VideoCapture camera(1);

	/*cpu memory*/
	unsigned char* cpu_sobel = new unsigned char[(width - 4 * grid_size.x) * (height - 4 * grid_size.y)];
	unsigned char* cpu_canny   = new unsigned char[(width - 4 * grid_size.x) * (height - 4 * grid_size.y)];

	/*gpu memory*/
	unsigned char* gpu_img;
	hipMalloc(&gpu_img, width * height * sizeof(unsigned char));

	unsigned char* gpu_sobel;
	hipMalloc(&gpu_sobel, (width - 4 * grid_size.x) * (height - 4 * grid_size.y) * sizeof(unsigned char));

	while (1)
	{
		camera >> img_src;
		resize(img_src, img_src, Size(width, height), 0, 0);
		cvtColor(img_src, img_src, CV_BGR2GRAY);
		imshow("img_src", img_src);

		hipMemcpy(gpu_img, img_src.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
		CUDA_GaussianAndSobel <<<grid_size, block_size, 3 * (BLOCK_SIZE_X) * (BLOCK_SIZE_Y) * sizeof(unsigned char) >> > (gpu_img, width, height, gpu_sobel);
		hipMemcpy(cpu_sobel, gpu_sobel, (width - 4 * grid_size.x) * (height - 4 * grid_size.y) * sizeof(unsigned char), hipMemcpyDeviceToHost);
		img_sobel = Mat(Size(width - 4 * grid_size.x, height - 4 * grid_size.y), CV_8UC1, cpu_sobel);
		imshow("img_guass&sobel", img_sobel);

		if ('q' == waitKey(1))
		{
			destroyAllWindows();

			free(cpu_sobel);
			cpu_sobel = NULL;
			free(cpu_canny);
			cpu_canny = NULL;

			hipFree(gpu_img);
			hipFree(gpu_sobel);

			break;
		}
	}
}

__global__ void CUDA_AssignTest(int* arr)
{
	int head = blockDim.x * blockDim.y * blockIdx.y * gridDim.x + blockIdx.x * blockDim.x;
	int id = head + blockDim.x * gridDim.x * threadIdx.y + threadIdx.x;
	int index = threadIdx.x + blockDim.x * threadIdx.y;

	extern __shared__ int data[];
	int* img = (int*)data;

	printf("[(%2d, %2d), (%2d,%2d), %2d]%2d\n", blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x, index, arr[id]);

	img[index] = arr[id] * 10000 + blockIdx.y * 1000 + blockIdx.x * 100 + threadIdx.y * 10 + threadIdx.x;
	arr[id] = img[index];
}

__global__ void CUDA_GaussianAndSobel(unsigned char* img, int width, int height, unsigned char* output)
{
	__shared__ unsigned char raw[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];
	__shared__ unsigned char gauss[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];
	__shared__ unsigned char sobel[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];

	int head = blockDim.x * blockDim.y * blockIdx.y * gridDim.x + blockIdx.x * blockDim.x;
	int id = head + blockDim.x * gridDim.x * threadIdx.y + threadIdx.x;
	int img_index = threadIdx.x + blockDim.x * threadIdx.y;

	if (id >= width * height)
		return;

	/*assign img to cache*/
	raw[img_index] = img[id];

	__syncthreads();

	/*guassian filter*/
	CUDA_Gaussian(raw, blockDim.x, blockDim.y, img_index, gauss);

	__syncthreads();

	/*sobel filter*/
	CUDA_Sobel(gauss, blockDim.x, blockDim.y, img_index, sobel);

	/*exclude block edge*/
	int i = img_index / blockDim.x;
	int j = img_index % blockDim.x;
	if (i <= 1 || i >= blockDim.y - 2 || j <= 1 || j >= blockDim.x - 2)
		return;
	int raw_head = blockIdx.y * (blockDim.x - 4) * (blockDim.y - 4) * gridDim.x + (i - 2) * (blockDim.x - 4) * gridDim.x;
	int col_bias = blockIdx.x * (blockDim.x - 4) + (j - 2);
	int new_id = raw_head + col_bias;

	output[new_id] = sobel[img_index];
}


__global__ void CUDA_GaussianAndSobel2(unsigned char* img, int width, int height, unsigned char* output)
{
	__shared__ unsigned char raw[(BLOCK_SIZE_X + 4) * (BLOCK_SIZE_Y + 4)];
	__shared__ unsigned char gauss[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];
	__shared__ unsigned char sobel[(BLOCK_SIZE_X) * (BLOCK_SIZE_Y)];

	int head = blockDim.x * blockDim.y * blockIdx.y * gridDim.x + blockIdx.x * blockDim.x;
	int img_index = head + blockDim.x * gridDim.x * threadIdx.y + threadIdx.x;                         //real index in image
	int index = threadIdx.x + blockDim.x * threadIdx.y;                                                //index in target block
	int extended_index = (index / BLOCK_SIZE_X + 2) * (BLOCK_SIZE_X + 4) + (index % BLOCK_SIZE_X + 2); //index in extended target block

	if (img_index >= width * height)
		return;

	/*assign img to cache*/
	raw[extended_index] = img[img_index];
	int i = index / width;
	int j = index % width;
	//left and right
	if (threadIdx.x == 0)
	{
		raw[extended_index - 1] = CUDA_GetPixelVal(img, width, height, i, j - 1);
		raw[extended_index - 2] = CUDA_GetPixelVal(img, width, height, i, j - 2);
	}
	else if (threadIdx.x == BLOCK_SIZE_X - 1)
	{
		raw[extended_index + 1] = CUDA_GetPixelVal(img, width, height, i, j + 1);
		raw[extended_index + 2] = CUDA_GetPixelVal(img, width, height, i, j + 2);
	}
	//top and bottom
	if (threadIdx.y == 0)
	{
		raw[extended_index - (BLOCK_SIZE_X + 4) * 1] = CUDA_GetPixelVal(img, width, height, i - 1, j);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 2] = CUDA_GetPixelVal(img, width, height, i - 2, j);
	}
	else if (threadIdx.y == BLOCK_SIZE_Y - 1)
	{
		raw[extended_index + (BLOCK_SIZE_X + 4) * 1] = CUDA_GetPixelVal(img, width, height, i + 1, j);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 2] = CUDA_GetPixelVal(img, width, height, i + 2, j);
	}
	//diagonal
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		raw[extended_index - (BLOCK_SIZE_X + 4) * 1 - 1] = CUDA_GetPixelVal(img, width, height, i - 1, j - 1);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 1 - 2] = CUDA_GetPixelVal(img, width, height, i - 1, j - 2);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 2 - 1] = CUDA_GetPixelVal(img, width, height, i - 2, j - 1);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 2 - 2] = CUDA_GetPixelVal(img, width, height, i - 2, j - 2);
	}
	else if (threadIdx.x == BLOCK_SIZE_X - 1 && threadIdx.y == 0)
	{
		raw[extended_index - (BLOCK_SIZE_X + 4) * 1 + 1] = CUDA_GetPixelVal(img, width, height, i - 1, j + 1);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 1 + 2] = CUDA_GetPixelVal(img, width, height, i - 1, j + 2);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 2 + 1] = CUDA_GetPixelVal(img, width, height, i - 2, j + 1);
		raw[extended_index - (BLOCK_SIZE_X + 4) * 2 + 2] = CUDA_GetPixelVal(img, width, height, i - 2, j + 2);
	}
	else if (threadIdx.x == 0 && threadIdx.y == BLOCK_SIZE_Y - 1)
	{
		raw[extended_index + (BLOCK_SIZE_X + 4) * 1 - 1] = CUDA_GetPixelVal(img, width, height, i + 1, j - 1);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 1 - 2] = CUDA_GetPixelVal(img, width, height, i + 1, j - 2);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 2 - 1] = CUDA_GetPixelVal(img, width, height, i + 2, j - 1);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 2 - 2] = CUDA_GetPixelVal(img, width, height, i + 2, j - 2);
	}
	else if (threadIdx.x == BLOCK_SIZE_X - 1 && threadIdx.y == BLOCK_SIZE_Y - 1)
	{
		raw[extended_index + (BLOCK_SIZE_X + 4) * 1 + 1] = CUDA_GetPixelVal(img, width, height, i + 1, j + 1);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 1 + 2] = CUDA_GetPixelVal(img, width, height, i + 1, j + 2);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 2 + 1] = CUDA_GetPixelVal(img, width, height, i + 2, j + 1);
		raw[extended_index + (BLOCK_SIZE_X + 4) * 2 + 2] = CUDA_GetPixelVal(img, width, height, i + 2, j + 2);
	}
	

	__syncthreads();

	output[img_index] = sobel[index];
}



__device__ void CUDA_Gaussian(unsigned char* img, int width, int height, int idx, unsigned char* output)
{
	int new_pixel_value = 0;
	int i = idx / width;
	int j = idx % width;
	new_pixel_value = CUDA_GetPixelVal(img, width, height, i - 1, j - 1) * 0.07511 +
					  CUDA_GetPixelVal(img, width, height, i - 1, j    ) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, i - 1, j + 1) * 0.07511 +
					  CUDA_GetPixelVal(img, width, height, i    , j - 1) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, i    , j    ) * 0.20418 +
					  CUDA_GetPixelVal(img, width, height, i    , j + 1) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, i + 1, j - 1) * 0.07511 +
					  CUDA_GetPixelVal(img, width, height, i + 1, j    ) * 0.12384 +
					  CUDA_GetPixelVal(img, width, height, i + 1, j + 1) * 0.07511;
	output[idx] = new_pixel_value;
}

__device__ void CUDA_Sobel(unsigned char* img, int width, int height, int idx, unsigned char* output)
{
	int sobel_x = 0;
	int sobel_y = 0;
	int sobel = 0;
	int i = idx / width;
	int j = idx % width;
	sobel_x = CUDA_GetPixelVal(img, width, height, i - 1, j - 1) * (1) +
			  CUDA_GetPixelVal(img, width, height, i - 1, j    ) * (2) +
			  CUDA_GetPixelVal(img, width, height, i - 1, j + 1) * (1) +
			  CUDA_GetPixelVal(img, width, height, i + 1, j - 1) * (-1) +
			  CUDA_GetPixelVal(img, width, height, i + 1, j    ) * (-2) +
			  CUDA_GetPixelVal(img, width, height, i + 1, j + 1) * (-1);
	sobel_y = CUDA_GetPixelVal(img, width, height, i - 1, j - 1) * (-1) +
		      CUDA_GetPixelVal(img, width, height, i - 1, j + 1) * (1) +
		      CUDA_GetPixelVal(img, width, height, i    , j - 1) * (-2) +
		      CUDA_GetPixelVal(img, width, height, i    , j + 1) * (2) +
		      CUDA_GetPixelVal(img, width, height, i + 1, j - 1) * (-1) +
		      CUDA_GetPixelVal(img, width, height, i + 1, j + 1) * (1);
	sobel = sqrtf((float)(sobel_x * sobel_x + sobel_y * sobel_y));
	if(sobel < 255)
		output[idx] = sobel;
	else
		output[idx] = 255;

	if (i <=1 || i >= height - 2)
		output[idx] = 0;
	if(j <= 1 || j >= width - 2)
		output[idx] = 0;
}

__device__ unsigned char CUDA_GetPixelVal(unsigned char* img, int width, int height, int i, int j)
{
	if (i >= height || i < 0)
		return 0;
	else if (j >= width || j < 0)
		return 0;
	return *(img + i * width + j);
}







void Test1()
{
	clock_t s_t, e_t;
	int N = 1920 * 1080 * 3;
	float* a = (float*)malloc(N * sizeof(float));
	float* b = (float*)malloc(N * sizeof(float));
	float* c = (float*)malloc(N * sizeof(float));

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * 0.01;
		c[i] = 0;
	}

	float* cuda_a;
	float* cuda_b;
	float* cuda_c;

	hipMalloc(&cuda_a, N * sizeof(float));
	hipMalloc(&cuda_b, N * sizeof(float));
	hipMalloc(&cuda_c, N * sizeof(float));

	hipMemcpy(cuda_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, N * sizeof(float), hipMemcpyHostToDevice);

	int thread_cnt = 1024;
	int block_cnt = (thread_cnt + N - 1) / thread_cnt;

	StopWatchInterface * timer_cublas;  //****��������GPU�˺�����ʱ
	sdkCreateTimer(&timer_cublas);		//****
	sdkStartTimer(&timer_cublas);		//****

	for (int i = 0; i < 100; i++)
	{
		VecAdd << <block_cnt, thread_cnt >> > (cuda_a, cuda_b, cuda_c, N);
		hipMemcpy(c, cuda_c, N * sizeof(float), hipMemcpyDeviceToHost);
	}

	hipDeviceSynchronize();			//****
	sdkStopTimer(&timer_cublas);		//****
	double dSeconds = sdkGetTimerValue(&timer_cublas) / (1000.0f); //***

	printf("\ngpu_done %.3f\n\n", dSeconds);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);


	LARGE_INTEGER litmp;	//####//��������cpu����ʱ��
	LONGLONG qt1, qt2;		//####
	double dft, dff, dfm;

	QueryPerformanceFrequency(&litmp);	//####���ʱ��Ƶ��
	dff = (double)litmp.QuadPart;			//####
	QueryPerformanceCounter(&litmp);	//####//��ó�ʼֵ
	qt1 = litmp.QuadPart;

	for (int i = 0; i < 100; i++)
	{
		Addvec(a, b, c, N);
	}

	QueryPerformanceCounter(&litmp);	//####//�����ֵֹ
	qt2 = litmp.QuadPart;					//####
	dfm = (double)(qt2 - qt1);				//####
	dft = dfm / dff;

	printf("\ncpu_done %.3f\n\n", dft);

	free(a);
	free(b);
	free(c);
}

void OpencvTest()
{
	int width  = 1920;
	int height = 1200;
	VideoCapture camera(0);
	Mat img_src, img_gray, img_sobel;

	float sobel_kernel_cpu[9] = { 1,1,1,0,0,0,-1,-1,-1 };
	unsigned char* sobel_cpu = new unsigned char[width * height];

	unsigned char* gray_gpu;
	unsigned char* sobel_gpu;
	float* sobel_kernel_gpu;
	hipMalloc(&gray_gpu, width * height * sizeof(unsigned char));
	hipMalloc(&sobel_gpu, width * height * sizeof(unsigned char));
	hipMalloc(&sobel_kernel_gpu, 9 * sizeof(float));
	hipMemcpy(sobel_kernel_gpu, sobel_kernel_cpu, 9 * sizeof(float), hipMemcpyHostToDevice);

	while (1)
	{
		camera >> img_src;
		imshow("img_src", img_src);
		resize(img_src, img_src, Size(width, height), 0, 0);
		cvtColor(img_src, img_gray, CV_BGR2GRAY);
		imshow("img_gray", img_gray);
		
		/*ConvCPU(img_gray.data, sobel_kernel_cpu, width, height, 3, sobel_cpu);
		ConvCPU(img_gray.data, sobel_kernel_cpu, width, height, 3, sobel_cpu);
		ConvCPU(img_gray.data, sobel_kernel_cpu, width, height, 3, sobel_cpu);
		ConvCPU(img_gray.data, sobel_kernel_cpu, width, height, 3, sobel_cpu);
		img_sobel = Mat(Size(width, height), CV_8UC1, sobel_cpu);
		imshow("sobel_cpu", img_sobel);*/

		int thread_size = 1024;
		int block_size = (width * height + thread_size - 1) / thread_size;
		hipMemcpy(gray_gpu, img_gray.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
		ConvGPU<<<block_size, thread_size>>>(gray_gpu, sobel_kernel_gpu, width, height, 3, sobel_gpu);
		hipMemcpy(sobel_cpu, sobel_gpu, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
		img_sobel = Mat(Size(width, height), CV_8UC1, sobel_cpu);
		imshow("sobel_cpu", img_sobel);

		if ('q' == waitKey(1))
		{
			destroyAllWindows();
			hipFree(gray_gpu);
			hipFree(sobel_gpu);
			hipFree(sobel_kernel_gpu);
			free(sobel_kernel_gpu);
			sobel_kernel_gpu = NULL;
			break;
		}
	}
}

void ConvCPU(unsigned char* img, float* kernel, int img_width, int img_height, int kernel_size, unsigned char* output)
{
	for (int i = 0; i < img_height; i++)
	{
		for (int j = 0; j < img_width; j++)
		{
			float conv_val = 0.0f;
			for (int k = 0; k < kernel_size; k++)
			{
				for (int m = 0; m < kernel_size; m++)
				{
					int pixel_i = k - kernel_size / 2 + i;
					int pixel_j = m - kernel_size / 2 + j;
					unsigned char pixel_val = 0;
					if (pixel_i < 0 || pixel_i >= img_height || pixel_j < 0 || pixel_j >= img_width)
					{

					}
					else
					{
						pixel_val = *(img + pixel_i * img_width + pixel_j);
					}
					int temp = pixel_val * (*(kernel + k * kernel_size + m));
					conv_val += temp;
				}
			}
			*(output + i * img_width + j) = fabs(conv_val);
		}
	}
}

__global__ void ConvGPU(unsigned char* img, float* kernel, int img_width, int img_height, int kernel_size, unsigned char* output)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = x / img_width;
	int j = x % img_width;

	if (x <= img_width * img_height - 1)
	{
		float conv_val = 0.0f;
		for (int k = 0; k < kernel_size; k++)
		{
			for (int m = 0; m < kernel_size; m++)
			{
				int pixel_i = k - kernel_size / 2 + i;
				int pixel_j = m - kernel_size / 2 + j;
				unsigned char pixel_val = 0;
				if (pixel_i < 0 || pixel_i >= img_height || pixel_j < 0 || pixel_j >= img_width)
				{

				}
				else
				{
					pixel_val = *(img + pixel_i * img_width + pixel_j);
				}
				int temp = pixel_val * (*(kernel + k * kernel_size + m));
				conv_val += temp;
			}
		}
		output[x] = fabs(conv_val);
	}
}

__global__ void ReverseGPU(unsigned char* img, int img_width, int img_height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x <= img_height * img_width - 1)
	{
		img[x] = 255 - img[x];
	}
}




void GpuArraySum()
{
	int arr[16];
	for (int i = 0; i < 16; i++)
	{
		arr[i] = i + 1;
	}

	int* cuda_arr;
	hipMalloc(&cuda_arr, 16 * sizeof(int));
	hipMemcpy(cuda_arr, arr, 16 * sizeof(int), hipMemcpyHostToDevice);

	int* cuda_result;
	hipMalloc(&cuda_result, 1 * sizeof(int));

	ArraySum<<<1, 16>>>(cuda_arr, cuda_result, 16);

	int result[1];
	hipMemcpy(result, cuda_result, 1 * sizeof(int), hipMemcpyDeviceToHost);

	printf("%d\n", result[0]);

	hipFree(cuda_arr);
	hipFree(cuda_result);

	system("pause");
}

__global__ void ArraySum(int* arr, int* res, int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int data[16];
	data[x] = arr[x];
	__syncthreads();

	for (int i = size / 2; i > 0; i /= 2)
	{
		if (x < i)
		{
			data[x] += data[x + i];
		}
		__syncthreads();
	}
	res[0] = data[0];
}