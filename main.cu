#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <opencv2/opencv.hpp>
using namespace cv;
using namespace std;

void GenerateGaussKernel(int size, float sigma, float* kernel);
unsigned char GetPixelVal(unsigned char* img, int img_height, int img_width, int i, int j);
void Gauss(unsigned char* img, int img_width, int img_height, float* kernel, int kernel_size, unsigned char* output);
void Sobel(unsigned char* img, int img_width, int img_height, short* sobel_x, short* sobel_y, unsigned char* output);
void NoneMaxSuppress(unsigned char* sobel, int sobel_width, int sobel_height, short* sobel_x, short* sobel_y, unsigned char* output);
void DoubleThreshold(unsigned char* sobel, int sobel_width, int sobel_height, unsigned char* canny);

__device__ unsigned char CUDA_GetPixelVal(unsigned char* img, int img_height, int img_width, int i, int j);
__global__ void CUDA_Gauss(unsigned char* img, int img_width, int img_height, float* kernel, int kernel_size, unsigned char* output);
__global__ void CUDA_Sobel(unsigned char* img, int img_width, int img_height, short* sobel_x, short* sobel_y, unsigned char* output);

int main(int argc, char** argv)
{
	int cpu_gpu = 1;
	//cout<<(int)memcmp(argv[1], "gpu", 3)<<" "<<(int)memcmp(argv[1], "cpu", 3)<<endl;

	cout<<"---canny kasoku!---"<<endl;
	
	int width = 1280;
	int height = 960;
	int gauss_kernel_size = 3;
	
	int thread_size = 1024;
	int block_size  = (width * height + thread_size - 1) / thread_size;
	
	/*****cpu memory*****/
	unsigned char* gauss = new unsigned char[width * height];

	float* gauss_kernel = new float[gauss_kernel_size * gauss_kernel_size];
	GenerateGaussKernel(gauss_kernel_size, 1, gauss_kernel);

	short* sobel_x = new short[width * height];
	short* sobel_y = new short[width * height];
	unsigned char* sobel = new unsigned char[width * height];

	/*****gpu memory*****/
	unsigned char* cuda_gray;
	hipMalloc(&cuda_gray, width * height * sizeof(unsigned char));

	unsigned char* cuda_gauss;
	hipMalloc(&cuda_gauss, width * height * sizeof(unsigned char));

	float* cuda_gauss_kernel;
	hipMalloc(&cuda_gauss_kernel, width * height * sizeof(float));
	hipMemcpy(cuda_gauss_kernel, gauss_kernel, gauss_kernel_size * gauss_kernel_size * sizeof(float), hipMemcpyHostToDevice);

	short* cuda_sobel_x;
	hipMalloc(&cuda_sobel_x, width * height * sizeof(short));
	
	short* cuda_sobel_y;
	hipMalloc(&cuda_sobel_y, width * height * sizeof(short));
	
	unsigned char* cuda_sobel;
	hipMalloc(&cuda_sobel, width * height * sizeof(unsigned char));
	
	while(1)
	{
		if(cpu_gpu == 0)
		{
			Mat img_src   = imread("/home/katsuto/Pictures/Wallpapers/timg.jpeg");
			Mat img_gray, img_gauss, img_sobel, img_canny;
			cvtColor(img_src, img_gray, CV_BGR2GRAY);
			
			resize(img_gray, img_gray, Size(width, height), 0, 0);
			imshow("img_gray", img_gray);
	
			Gauss(img_gray.data, width, height, gauss_kernel, gauss_kernel_size, gauss);
			//img_gauss = Mat(Size(width, height), CV_8UC1, gauss);
			//imshow("img_gauss", img_gauss);
		
			Sobel(gauss, width, height, sobel_x, sobel_y, sobel);
			img_sobel = Mat(Size(width, height), CV_8UC1, sobel);
			imshow("img_sobel", img_sobel);
		}
		else
		{
			/*read image*/
			Mat img_src   = imread("/home/katsuto/Pictures/Wallpapers/timg.jpeg");
			Mat img_gray, img_gauss, img_sobel, img_canny;
			cvtColor(img_src, img_gray, CV_BGR2GRAY);
			resize(img_gray, img_gray, Size(width, height), 0, 0);
			imshow("img_gray", img_gray);
			
			/*load into gpu*/
			hipMemcpy(cuda_gray, img_gray.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
			
			/*gauss filter*/
			CUDA_Gauss<<<block_size, thread_size>>>(cuda_gray, width, height, cuda_gauss_kernel, gauss_kernel_size, cuda_gauss);
			//hipMemcpy(gauss, cuda_gauss, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
			//img_gauss = Mat(Size(width, height), CV_8UC1, gauss);
			//imshow("img_gauss_cuda", img_gauss);
		
			/*sobel edge detection*/
			CUDA_Sobel<<<block_size, thread_size>>>(cuda_gauss, width, height, cuda_sobel_x, cuda_sobel_y, cuda_sobel);
			hipMemcpy(sobel, cuda_sobel, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
			img_sobel = Mat(Size(width, height), CV_8UC1, sobel);
			imshow("img_sobel_gpu", img_sobel);

		}
		waitKey(0);
		break;
	}

	hipFree(cuda_gray);
	hipFree(cuda_gauss);
	hipFree(cuda_gauss_kernel);
	hipFree(cuda_sobel_x);
	hipFree(cuda_sobel_y);
	hipFree(cuda_sobel);

	delete[] gauss;
	gauss = nullptr;
	delete[] gauss_kernel;
	gauss_kernel = nullptr;
	delete[] sobel_x;
	sobel_x = nullptr;

	delete[] sobel_y;
	sobel_y = nullptr;
	delete[] sobel;
	sobel = nullptr;

	return 0;
}

unsigned char GetPixelVal(unsigned char* img, int img_height, int img_width, int i, int j)
{
	if(i >= img_height || i < 0)
		return 0;
	if(j >= img_width  || j < 0)
		return 0;
	return *(img + i * img_width + j);	
}

__device__ unsigned char CUDA_GetPixelVal(unsigned char* img, int img_height, int img_width, int i, int j)
{	
	if(i >= img_height || i < 0)
		return 0;	
	else if(j >= img_width  || j < 0)
		return 0;
	return *(img + i * img_width + j);	
}

__global__ void CUDA_Gauss(unsigned char* img, int img_width, int img_height, float* kernel, int kernel_size, unsigned char* output)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int i  = id / img_width;
	int j  = id % img_width;
	if(id < img_width * img_height)
	{
		int new_pixel_value  = 0;
		int half_kernel_size = kernel_size / 2;
		for(int k = 0; k < kernel_size; k++)
		{
			for(int m = 0; m < kernel_size; m++)
			{
				new_pixel_value += (*(kernel + k * kernel_size + m)) * CUDA_GetPixelVal(img, img_height, img_width, i + k - half_kernel_size, j + m - half_kernel_size);
				__syncthreads();
			}
		}	
		*(output + i * img_width + j) = new_pixel_value;
	}
}

__global__ void CUDA_Sobel(unsigned char* img, int img_width, int img_height, short* sobel_x, short* sobel_y, unsigned char* output)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int i = id / img_width;
	int j = id % img_width;
	
	if(id >= img_width * img_height)
		return;

	*(sobel_x + i * img_width + j) = CUDA_GetPixelVal(img, img_height, img_width, i-1, j-1) * (1) +
			           		  		 CUDA_GetPixelVal(img, img_height, img_width, i-1, j  ) * (2) +
	                        	     CUDA_GetPixelVal(img, img_height, img_width, i-1, j+1) * (1) +
	                           	     CUDA_GetPixelVal(img, img_height, img_width, i  , j-1) * (0) +
	                             	 CUDA_GetPixelVal(img, img_height, img_width, i  , j  ) * (0) +
	                                 CUDA_GetPixelVal(img, img_height, img_width, i  , j+1) * (0) +
	                                 CUDA_GetPixelVal(img, img_height, img_width, i+1, j-1) * (-1) +
	                                 CUDA_GetPixelVal(img, img_height, img_width, i+1, j  ) * (-2) +
	                                 CUDA_GetPixelVal(img, img_height, img_width, i+1, j+1) * (-1);


	*(sobel_y + i * img_width + j) = CUDA_GetPixelVal(img, img_height, img_width, i-1, j-1) * (-1) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i-1, j  ) * (0) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i-1, j+1) * (1) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i  , j-1) * (-2) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i  , j  ) * (0) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i  , j+1) * (2) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i+1, j-1) * (-1) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i+1, j  ) * (0) +
		                             CUDA_GetPixelVal(img, img_height, img_width, i+1, j+1) * (1);

	float val =sqrt(pow(*(sobel_x + i * img_width + j), 2) + pow(*(sobel_y + i * img_width + j), 2));
	if(val > 255)
		*(output + i * img_width + j) = 255;
	else
		*(output + i * img_width + j) = val;	
}

void GenerateGaussKernel(int size, float sigma, float* kernel)
{
	int center = size / 2;
	float sum = 0.0f;
	for(int i = 0; i < size; i++)
	{
		for(int j = 0; j < size; j++)
		{
			*(kernel + i * size + j) = (float)1 / (2 * 3.1415926 * sigma * sigma) * exp(-(pow(i - center, 2) + pow(j - center, 2)) / (2 * pow(sigma, 2)));
			sum += *(kernel + i * size + j);
		}
	}	
	cout<<"gauss kenel : "<<endl;
	for(int i = 0; i < size; i++)
	{
		for(int j = 0; j < size; j++)
		{
			*(kernel + i * size + j) /= sum;
			cout<<*(kernel + i * size + j)<<" ";
		}
		cout<<endl;
	}
	cout<<endl;
}

void Gauss(unsigned char* img, int img_width, int img_height, float* kernel, int kernel_size, unsigned char* output)
{
	for(int i = 0; i < img_height; i++)
	{
		for(int j = 0; j < img_width; j++)
		{
			int new_pixel_value  = 0;
			int half_kernel_size = kernel_size / 2;
			for(int k = 0; k < kernel_size; k++)
			{
				for(int m = 0; m < kernel_size; m++)
				{
					new_pixel_value += GetPixelVal(img, img_height, img_width, i + k - half_kernel_size, j + m - half_kernel_size) * (*(kernel + k * kernel_size + m));
				}
			}
			*(output + i * img_width + j) = new_pixel_value;
		}
	}
}

void Sobel(unsigned char* img, int img_width, int img_height, short* sobel_x, short* sobel_y, unsigned char* output)
{
	float sobel_filter_x[9] = {1,2,1,0,0,0,-1,-2,-1};
	float sobel_filter_y[9] = {-1,0,1,-2,0,2,-1,0,1};
	
	for(int i = 0; i < img_height; i++)
	{
		for(int j = 0; j < img_width; j++)
		{
			*(sobel_x + i * img_width + j) = GetPixelVal(img, img_height, img_width, i-1, j-1) * sobel_filter_x[0] +
					           		  		 GetPixelVal(img, img_height, img_width, i-1, j  ) * sobel_filter_x[1] +
			                        	     GetPixelVal(img, img_height, img_width, i-1, j+1) * sobel_filter_x[2] +
			                           	     GetPixelVal(img, img_height, img_width, i  , j-1) * sobel_filter_x[3] +
			                             	 GetPixelVal(img, img_height, img_width, i  , j  ) * sobel_filter_x[4] +
			                                 GetPixelVal(img, img_height, img_width, i  , j+1) * sobel_filter_x[5] +
			                                 GetPixelVal(img, img_height, img_width, i+1, j-1) * sobel_filter_x[6] +
			                                 GetPixelVal(img, img_height, img_width, i+1, j  ) * sobel_filter_x[7] +
			                                 GetPixelVal(img, img_height, img_width, i+1, j+1) * sobel_filter_x[8];


			*(sobel_y + i * img_width + j) = GetPixelVal(img, img_height, img_width, i-1, j-1) * sobel_filter_y[0] +
				                             GetPixelVal(img, img_height, img_width, i-1, j  ) * sobel_filter_y[1] +
				                             GetPixelVal(img, img_height, img_width, i-1, j+1) * sobel_filter_y[2] +
				                             GetPixelVal(img, img_height, img_width, i  , j-1) * sobel_filter_y[3] +
				                             GetPixelVal(img, img_height, img_width, i  , j  ) * sobel_filter_y[4] +
				                             GetPixelVal(img, img_height, img_width, i  , j+1) * sobel_filter_y[5] +
				                             GetPixelVal(img, img_height, img_width, i+1, j-1) * sobel_filter_y[6] +
				                             GetPixelVal(img, img_height, img_width, i+1, j  ) * sobel_filter_y[7] +
				                             GetPixelVal(img, img_height, img_width, i+1, j+1) * sobel_filter_y[8];

			float val =sqrt(pow(*(sobel_x + i * img_width + j), 2) + pow(*(sobel_y + i * img_width + j), 2));
			if(val > 255)
				*(output + i * img_width + j) = 255;
			else	
				*(output + i * img_width + j) = val;
		}
	}
}

void NoneMaxSuppress(unsigned char* sobel, int sobel_width, int sobel_height, short* sobel_x, short* sobel_y, unsigned char* output)
{
	
}

void DoubleThreshold(unsigned char* sobel, int sobel_width, int sobel_height, unsigned char* canny)
{
	
}
